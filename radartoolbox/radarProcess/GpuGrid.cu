#include "hip/hip_runtime.h"
//
#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include ""
#include <hipfft/hipfft.h>
#include <stdlib.h>
#include "GpuGrid.h"

//#include <cuPrintf.cu>
#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>
#ifndef _SIMPLETEXTURE_KERNEL_H_
#define _SIMPLETEXTURE_KERNEL_H_
#define PI 3.14159265
#define SPEED_OF_LIGHT 299702547

texture<float, hipTextureType2D, hipReadModeElementType > texRefPolar; //hipReadModeNormalizedFloat
texture<float, hipTextureType1D,  hipReadModeElementType> texRefAzimuth;



__global__ void interpPolerKernel(float* output, int width, int height,short *azimuthMask) 
{ // Calculate normalized texture coordinates 
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y; 
	if ((y<width) && (x<height)) {
			float angle = tex1D(texRefAzimuth, ((float)x));
			 //tex2D(texRefPolar, ((float)y),angle);tex2D(texRefPolar, ((float)y),angle) *
			 output[x * width + y] = tex2D(texRefPolar, ((float)y),angle)*((float) azimuthMask[x]); //(float)x;//tex2D(texRef, tu, tv); 
	}
//	output[y*width + x] = tex2D(texRefPolar, (float)x, (float)y) ;
}

__global__ void interpCartKernel(float* output, int width, int height, float antennaHieght,int xOffset,int yOffset, float heading, float gridSize, float rangeBinSize, float angleStepInRadians,float maxrange,bool tvg, int maxAzimuthBin) 
{ // Calculate normalized texture coordinates 
	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y; 
	if ((idy>0) && (idy<width) && (idx<height) && (idx>0)) {
		float xdistance = (xOffset-((float)height-(float)idx))*gridSize;
        float ydistance =(yOffset-(float)idy)*gridSize;		
        float azimuth =atan2(ydistance,xdistance)-heading;
		azimuth =2*PI-(azimuth + ((azimuth<0)*PI*2)); // convert to continous positive
		azimuth = (azimuth>2*PI)*(azimuth-2*PI) + (azimuth<2*PI)*(azimuth);
		azimuth=(azimuth)/angleStepInRadians;
		float azimuth1D= tex1D(texRefAzimuth, (azimuth)); // interp the bin
        float range = sqrt((xdistance*xdistance)+(ydistance*ydistance));
		range = sqrt((antennaHieght*antennaHieght)+(range*range))/rangeBinSize; // allow for the height of the antenna
		output[((height-1)*(width))-(idx * width) + idy] = tex2D(texRefPolar, range,azimuth1D)*10 * (float)(range<maxrange) * (float) (azimuth<maxAzimuthBin); //* ((float)(tvg)*range*rangeBinSize/1000); //(float)x;//tex2D(texRef, tu, tv); 

	}
}

__global__ void gpuinterpolateAngles(float *output, float *frameAngle, int numberOfAngleBins, float angleStep) 
 {
	  unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	  if (idx<360/angleStep) // initate the output
		  output[idx]=-9999;
		  int startbin = 1;
	  if (idx==0)
		  output[0] =0;
	  else if (idx<numberOfAngleBins) {
		  float x0 =frameAngle[idx-1];
		  float x1 = frameAngle[idx];
		  int startbin =(x0/angleStep)+1;
		  int endbin =	(x1/angleStep);
		  for (int i=startbin;i<=endbin;i++){
			 float x = (float)i*angleStep;
			 output[i] = idx-1 +(x-x0)*((1)/(x1-x0));
		  }
	  }

 }



 GpuGrid::GpuGrid(int collections, int rangeBins, float step,int xSize,int ySize,float sampleRate, bool applyTvg) 
{
	numberOfCollectionsPerRotation =collections;
	numberOfRangeBins =rangeBins;
	rangeBinSize = SPEED_OF_LIGHT/(sampleRate*1000000)/2;
	angleStep = step;
	numberOfAngleBins= 360/angleStep;
	xOutputSize = xSize;
	yOutputSize = ySize;
	tvg = applyTvg;
	// allocate memory on the GPU
	hipHostMalloc((void**)&rawFrameBuffer, numberOfCollectionsPerRotation*numberOfRangeBins*sizeof(float));
	hipHostMalloc((void**)&rawAngleBuffer, numberOfCollectionsPerRotation*sizeof(float));
	hipHostMalloc((void**)&interplatedPolarFrame, sizeof(float)*xOutputSize*yOutputSize);
    hipHostMalloc((void**)&azimuthArray,  numberOfAngleBins * sizeof(float));

	// allocate paged locked memory on the host
	hipMalloc((void**)&deviceRawAngleBuffer, numberOfCollectionsPerRotation*sizeof(float));
	hipMalloc((void**)&deviceAzimuth, numberOfAngleBins*sizeof(float));
	hipMalloc((void**)&deviceFrameBuffer, xOutputSize*yOutputSize*sizeof(float));

	channelDescPolar = hipCreateChannelDesc<float>();
	channelDescAzimuth = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
	hipMallocArray(&deviceRawFrameArray, &channelDescPolar, numberOfRangeBins, numberOfCollectionsPerRotation);
	hipMallocArray(&deviceAzimuthArray, &channelDescAzimuth, numberOfAngleBins, 0);

	selectedCudaDeviceId =initGPU()>0;
	gpuReady =selectedCudaDeviceId>-1;

}

  GpuGrid::~GpuGrid()
{
	// free the Host stuff
	hipHostFree(rawFrameBuffer);
	hipHostFree(rawAngleBuffer);
	hipHostFree(interplatedPolarFrame);
    hipHostFree(azimuthArray);

	// free device memory
	hipFree(deviceRawAngleBuffer);
	hipFree(deviceFrameBuffer);
	hipFree(deviceAzimuth);
	

	// Free Cuda arrays
	hipFreeArray(deviceRawFrameArray);
	hipFreeArray(deviceAzimuthArray);

}

 void GpuGrid::cpyFrame(short *source)
 {
	// memcpy(rawFrameBuffer,source,numberOfCollectionsPerRotation*numberOfRangeBins*sizeof(short));
}

void GpuGrid::cpyAngles(float *source){
	// memcpy(rawAngleBuffer,source,numberOfCollectionsPerRotation*sizeof(float));
}

 void GpuGrid::interpolateAngles(float *frameAngle)
 {
/*	 float *devptr;
	  float y0,y1,x,x0,x1;
	  int sumCount =1;
	  azimuthArray[0] =0;
	  for (int i=1;(i<numberOfAngleBins);i++) {
		  x = (float)i*angleStep;
		  while ((x>frameAngle[sumCount]) && (sumCount<numberOfCollectionsPerRotation))
		    sumCount++;
		  if(sumCount<numberOfCollectionsPerRotation) {
			x0 = frameAngle[sumCount-1];
			x1 = frameAngle[sumCount];
			y0 = sumCount-1;
			y1 = sumCount;
			azimuthArray[i] = y0 +(x-x0)*((y1-y0)/(x1-x0));
			maxAngleBin = i;
		  }
	  }

	//   cudaPrintfInit(); */
 	  dim3 dimBlock(32, 1);  
	  dim3 dimGrid((numberOfCollectionsPerRotation + dimBlock.x - 1) / dimBlock.x,1);
	  maxAngleBin = rawAngleBuffer[numberOfCollectionsPerRotation-1]/angleStep;
	  hipMemcpy(deviceRawAngleBuffer,rawAngleBuffer, numberOfCollectionsPerRotation *sizeof(float), hipMemcpyHostToDevice);
	//  hipMemcpy(deviceAzimuth,azimuthArray, numberOfAngleBins *sizeof(float), hipMemcpyHostToDevice);
	  gpuinterpolateAngles<<<dimGrid, dimBlock>>>(deviceAzimuth,deviceRawAngleBuffer,numberOfCollectionsPerRotation,angleStep);
//	  cudaPrintfDisplay(stdout, true);
	//	cudaPrintfEnd();

 }



 
 float *GpuGrid::interpolatePolarFrame(float *sourceFrame,float *collectionAngles)
 {
	  if (gpuReady) {
		  interpolateAngles(collectionAngles);
		// store polar frame on GPU
		hipMemcpyToArray(deviceRawFrameArray, 0, 0, sourceFrame, numberOfCollectionsPerRotation*numberOfRangeBins*sizeof(float), hipMemcpyHostToDevice);
		//hipMemcpyToArray(deviceAzimuthArray, 0, 0, azimuthArray,numberOfAngleBins *sizeof(float), hipMemcpyHostToDevice);
	 //   hipMemcpy(deviceAzimuthMask, azimuthMask, numberOfAngleBins*sizeof(short), hipMemcpyHostToDevice);
		// Set texture parameters 
		texRefPolar.addressMode[0] = hipAddressModeClamp; 
		texRefPolar.addressMode[1] = hipAddressModeClamp; 
		texRefPolar.filterMode = hipFilterModeLinear; 
		texRefPolar.normalized = false;
		hipBindTextureToArray(&texRefPolar, deviceRawFrameArray,&channelDescPolar);

		//Set texture parameters Azimuth
		texRefAzimuth.addressMode[0] = hipAddressModeClamp; 
		texRefAzimuth.addressMode[1] = hipAddressModeClamp; 
		texRefAzimuth.filterMode = hipFilterModeLinear; 
		texRefAzimuth.normalized = false;
		hipBindTextureToArray(&texRefAzimuth, deviceAzimuthArray,&channelDescAzimuth);


	dim3 dimBlock(4, 4); 
	dim3 dimGrid((numberOfAngleBins + dimBlock.x - 1) / dimBlock.x, (numberOfRangeBins + dimBlock.y - 1) / dimBlock.y); 
//	interpPolerKernel<<<dimGrid, dimBlock>>>(deviceFrameBuffer, numberOfRangeBins,numberOfAngleBins,deviceAzimuthMask,tvg,maxAngleBin);
//	hipMemcpy(interplatedPolarFrame, deviceFrameBuffer, numberOfAngleBins*numberOfRangeBins*sizeof(float), hipMemcpyDeviceToHost);
	hipUnbindTexture( texRefPolar );
	hipUnbindTexture( texRefAzimuth );
	return(interplatedPolarFrame);
	  }
	  return(NULL);
 }


float *GpuGrid::interpolateCartFrame(float *sourceFrame,float *collectionAngles,int xOffset, int yOffset, float gridSize, float heading)
 {
	  if (gpuReady) {
	    interpolateAngles(collectionAngles);
		// store polar frame on GPU
	//    hipMemcpy(deviceAzimuthMask, azimuthMask, numberOfAngleBins*sizeof(short), hipMemcpyHostToDevice);
		hipMemcpyToArray(deviceRawFrameArray, 0, 0, sourceFrame, numberOfCollectionsPerRotation*numberOfRangeBins*sizeof(float), hipMemcpyHostToDevice);
		hipMemcpyToArray(deviceAzimuthArray, 0, 0, deviceAzimuth,numberOfAngleBins *sizeof(float), hipMemcpyDeviceToDevice);

		// Set texture parameters 
		texRefPolar.addressMode[0] = hipAddressModeClamp; 
		texRefPolar.addressMode[1] = hipAddressModeClamp; 
		texRefPolar.filterMode = hipFilterModeLinear; 
		texRefPolar.normalized = false;
		hipBindTextureToArray(&texRefPolar, deviceRawFrameArray,&channelDescPolar);

		//Set texture parameters Azimuth
		texRefAzimuth.addressMode[0] = hipAddressModeClamp; 
		texRefAzimuth.addressMode[1] = hipAddressModeClamp; 
		texRefAzimuth.filterMode = hipFilterModeLinear; 
		texRefAzimuth.normalized = false;
		hipBindTextureToArray(&texRefAzimuth, deviceAzimuthArray,&channelDescAzimuth);
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, selectedCudaDeviceId);
	dim3 dimBlock(16, 16); 
	int xgrid = (xOutputSize + dimBlock.x - 1) / dimBlock.x;
	int ygrid = (yOutputSize + dimBlock.y - 1) / dimBlock.y;
	if (xgrid<ygrid){
		if (ygrid % prop.multiProcessorCount!=0)
			ygrid = ygrid +prop.multiProcessorCount- ygrid % prop.multiProcessorCount;
	}else{
		if (xgrid % prop.multiProcessorCount!=0)
			xgrid = xgrid +prop.multiProcessorCount- xgrid % prop.multiProcessorCount;
	}
	dim3 dimGrid(xgrid,ygrid); 
	//convert heading to radians
	heading = PI*heading/180; 
	interpCartKernel<<<dimGrid, dimBlock>>>(deviceFrameBuffer, yOutputSize,xOutputSize,14.0,xOffset,yOffset,heading-(PI/2),gridSize,rangeBinSize,(PI*angleStep)/180,numberOfRangeBins,tvg,maxAngleBin);
	hipMemcpy(interplatedPolarFrame, deviceFrameBuffer, yOutputSize*xOutputSize*sizeof(float), hipMemcpyDeviceToHost);
	hipUnbindTexture( texRefPolar );
	hipUnbindTexture( texRefAzimuth );
	return(interplatedPolarFrame);
	  }
	  return(NULL);
 }





int GpuGrid::initGPU()
{
	hipDeviceProp_t prop;
	bool result = false;
	int dCnt = 0;
	int selectedCudaDeviceId = 0;
	hipGetDeviceCount(&dCnt);
	printf("number of cuda gpu devices: %d\n", dCnt);
	if (dCnt > 0) {
		if (dCnt > 1) {
			int multiprocessor_cnt = 0;

			for (int deviceId=0; deviceId<dCnt; ++deviceId) {
				if (hipSuccess == hipGetDeviceProperties(&prop, deviceId)) {
					if (prop.multiProcessorCount > multiprocessor_cnt) {
						multiprocessor_cnt = prop.multiProcessorCount;
						selectedCudaDeviceId = deviceId;
					}
				}
			}
		} else {
			selectedCudaDeviceId = 0;
		}
	printf("selected device with most multiprocessors: %d\n", selectedCudaDeviceId);
		hipSetDevice(selectedCudaDeviceId);
		hipGetDeviceProperties(&prop, selectedCudaDeviceId);
		return(selectedCudaDeviceId);
	}
	return (-1);
}
#endif // #ifndef _SIMPLETEXTURE_KERNEL_